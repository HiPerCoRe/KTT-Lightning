#include "hip/hip_runtime.h"
template <typename T, typename Operator>
__global__ void transform(const T* input1, const T* input2, T* out, const size_t numItems)
{
    Operator op;

    #pragma unroll UNROLL_FACTOR
    for (int element = 0; element < ELEMENTS_PER_THREAD; ++element)
    {
        #ifdef MEMORY_MAPPING_PATTERN == 0
        const size_t index = ELEMENTS_PER_THREAD * (blockIdx.x * blockDim.x + threadIdx.x);
        const size_t offsetIndex = index + element;
        #elif MEMORY_MAPPING_PATTERN == 1
        const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
        const size_t offsetIndex = index + ELEMENTS_PER_THREAD * blockDim.x * element;
        #endif

        if (offsetIndex < numItems)
        {
            out[offsetIndex] = op(input1[offsetIndex], input2[offsetIndex]);
        }
    }
};
