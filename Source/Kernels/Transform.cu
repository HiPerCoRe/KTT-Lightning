#include "hip/hip_runtime.h"
template <typename T, typename Operator>
__global__ void transform(const T* input1, const T* input2, T* out, const size_t numItems)
{
    Operator op;

    #pragma unroll UNROLL_FACTOR
    for (int element = 0; element < ELEMENTS_PER_THREAD; ++element)
    {
        const size_t index = ELEMENTS_PER_THREAD * blockIdx.x * blockDim.x + ELEMENTS_PER_THREAD * threadIdx.x;
        const size_t offsetIndex = index + element;

        if (offsetIndex < numItems)
        {
            out[offsetIndex] = op(input1[offsetIndex], input2[offsetIndex]);
        }
    }
};
